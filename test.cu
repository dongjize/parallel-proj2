#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc.hpp>
#include <time.h>
#include <iostream>

using namespace std;
using namespace cv;


int main() {
    clock_t t1 = clock();

    int num_devices = cv::cuda::getCudaEnabledDeviceCount();

    if (num_devices <= 0) {
        cerr << "There is no device." << endl;
        return -1;
    }

    int enable_device_id = -1;
    for (int i = 0; i < num_devices; i++) {
        cv::cuda::DeviceInfo dev_info(i);
        if (dev_info.isCompatible()) {
            enable_device_id = i;
        }
    }

    if (enable_device_id < 0) {
        cerr << "GPU module isn't built for GPU" << endl;
    }

    cv::cuda::setDevice(enable_device_id);

    Mat src_image = imread("Hi-RGB0971.tif");
    Mat dst_image;
    cuda::GpuMat d_src_img(src_image);//upload src image to gpu
    cuda::GpuMat d_dst_img;
    cuda::cvtColor(d_src_img, d_dst_img, CV_BGR2GRAY);
    d_dst_img.download(dst_image);
    imshow("test", dst_image);

    clock_t t2 = clock();

    std::cout << "time: " << (t2 - t1) / 1000 << "s" << endl;

    waitKey();

    return 0;

}