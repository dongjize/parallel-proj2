#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include "opencv2/core.hpp"
#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudafilters.hpp"
#include <iostream>

using namespace std;
using namespace cv;

__global__ void sobelInCuda(unsigned char *dataIn, unsigned char *dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    int Gx = 0;
    int Gy = 0;

    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth + xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
             - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) * imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
             - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) * imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        dataOut[index] = (abs(Gx) + abs(Gy)) / 2;
    }
}

void sobel(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    int Gx = 0;
    int Gy = 0;
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar *dataUp = srcImg.ptr<uchar>(i - 1);
        uchar *data = srcImg.ptr<uchar>(i);
        uchar *dataDown = srcImg.ptr<uchar>(i + 1);
        uchar *out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            out[j] = (abs(Gx) + abs(Gy)) / 2;
        }
    }
}

int main()
{
    Mat grayImg = imread("001.jpg", 0);

    int imgHeight = grayImg.rows;
    int imgWidth = grayImg.cols;

    Mat gaussImg;
    GaussianBlur(grayImg, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);

    Mat dst(imgHeight, imgWidth, CV_8UC1, Scalar(0));
    sobel(gaussImg, dst, imgHeight, imgWidth);

    Mat dstImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    unsigned char *d_in;
    unsigned char *d_out;

    hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));

    hipMemcpy(d_in, gaussImg.data, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    sobelInCuda << <blocksPerGrid, threadsPerBlock >> >(d_in, d_out, imgHeight, imgWidth);

    hipMemcpy(dstImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
