// driver for rsa cracking

#include <stdio.h>

#include "common.h"
#include "gcd.h"

/**
 * Main function to read keys from file and then matrix
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char *argv[]) {
    char *res, *cudaRes;
    bitInt *numbers, *cudaNums;

    if (argc != 2) {
        printf("error, syntax is %s <file name>\n", argv[0]);
        return 0;
    }

    int numKeys = readFile(argv[1], &numbers, &res);
    int countBytes = 1 + ((numKeys - 1) / 8);
    int numSize = SIZE_BYTES * numKeys;

    hipMalloc(&cudaNums, numSize);
    hipMalloc(&cudaRes, numKeys * countBytes);
    hipMemset(cudaRes, 0, numKeys * countBytes);

    hipMemcpy(cudaNums, numbers, numSize, hipMemcpyHostToDevice);

    int dimBlock = SIZE * 2; // the size of each block
    int dimGrid = 1 + ((numKeys - 1) / dimBlock); // the number of blocks divided into on the GPU

    printf("%d blocks of size %d\n", dimGrid, dimBlock);

    //Lets gcd
//    for (int offset = 0; offset < numKeys; offset += WORK_SIZE) {
//        findGCDs << < dimGrid, dimBlock >> > (cudaNums, numKeys, cudaRes, offset);
//    }
    gmpGCDs << < dimGrid, dimBlock >> > (cudaNums, numKeys, cudaRes);

    hipMemcpy(res, cudaRes, numKeys * countBytes, hipMemcpyDeviceToHost);

    hipFree(cudaNums);
    hipFree(cudaRes);

    writeFiles("privateKeys", numKeys, numbers, res);

    free(numbers);
    free(res);

    return 0;
}
