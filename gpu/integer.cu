#include "hip/hip_runtime.h"
#include "integer.h"

#include <string.h>
#include <stdio.h>
#include <stdlib.h>

__device__ void gcd(volatile uint32_t *x, volatile uint32_t *y);

__device__ void shiftR1(volatile uint32_t *x);

__device__ void shiftL1(volatile uint32_t *x);

__device__ int geq(volatile uint32_t *x, volatile uint32_t *y);

__device__ void cuSubtract(volatile uint32_t *x, volatile uint32_t *y, volatile uint32_t *z);

/**
 * See "PARIS: A Parallel RSA-Prime Inspection Tool" by Joseph White
 */
__global__ void
cuda_factorKeys(const integer *keys, uint16_t *notCoprime,
                int tileRow, int tileCol,
                int tileDim, int numKeys) {
    /**
     * shared memory for keys
     * each block has 16 warps, for 16 gcd calculations
     * create 2 blocks for shared memory with same dimensions as block size to store the keys in
     */
    __shared__
    volatile uint32_t y[BLOCK_DIM][BLOCK_DIM][32];
    __shared__
    volatile uint32_t z[BLOCK_DIM][BLOCK_DIM][32];

    /* determine key indexes */
    int keyX = tileCol * tileDim + blockIdx.x * BLOCK_DIM + threadIdx.y;
    int keyY = tileRow * tileDim + blockIdx.y * BLOCK_DIM + threadIdx.z;

    /* only continue w/ warp if we need to to run this comparison */
    if (keyX < numKeys && keyY < numKeys && keyX > keyY) {
        /* each thread loads its corresponding int into shared memory */
        y[threadIdx.y][threadIdx.z][threadIdx.x] = keys[keyX].ints[threadIdx.x];
        z[threadIdx.y][threadIdx.z][threadIdx.x] = keys[keyY].ints[threadIdx.x];

        /* run gcd */
        gcd(y[threadIdx.y][threadIdx.z], z[threadIdx.y][threadIdx.z]);

        if (threadIdx.x == 31) {
            /* turn gcd=1 to 0 */
            z[threadIdx.y][threadIdx.z][threadIdx.x] -= 1;

            /* check if any ints in the warp's shared memory are > 0, which means gcd > 1
             * update notCoprime */
            if (__any(z[threadIdx.y][threadIdx.z][threadIdx.x])) {
                int notCoprimeBlockNdx = blockIdx.y * gridDim.x + blockIdx.x;
                notCoprime[notCoprimeBlockNdx] |= 1 << threadIdx.z * BLOCK_DIM + threadIdx.y;
            }
        }
    }
}

void cudaWrapper(dim3 gridDim, dim3 blockDim, integer *d_keys, uint16_t *d_notCoprime,
                 int tileRow, int tileCol, int tileDim, int numKeys) {
    cuda_factorKeys << < gridDim, blockDim >> > (d_keys, d_notCoprime, tileRow, tileCol, tileDim, numKeys);
}

/**
 * Binary GCD algo
 */
__device__ void gcd(volatile uint32_t *x, volatile uint32_t *y) {
    int tid = threadIdx.x;

    while (__any(x[tid])) {
        while ((x[31] & 1) == 0)
            shiftR1(x);

        while ((y[31] & 1) == 0)
            shiftR1(y);

        if (geq(x, y)) {
            cuSubtract(x, y, x);
            shiftR1(x);
        } else {
            cuSubtract(y, x, y);
            shiftR1(y);
        }
    }
}

__device__ void shiftR1(volatile uint32_t *x) {
    int tid = threadIdx.x;
    uint32_t prevX = tid ? x[tid - 1] : 0;
    x[tid] = (x[tid] >> 1) | (prevX << 31);
}

__device__ void shiftL1(volatile uint32_t *x) {
    int tid = threadIdx.x;
    uint32_t nextX = tid != 31 ? x[tid + 1] : 0;
    x[tid] = (x[tid] << 1) | (nextX >> 31);
}

__device__ int geq(volatile uint32_t *x, volatile uint32_t *y) {
    /* shared memory to hold the position at which the int of x >= int of y */
    __shared__ unsigned int pos[BLOCK_DIM][BLOCK_DIM];
    int tid = threadIdx.x;

    if (tid == 0)
        pos[threadIdx.y][threadIdx.z] = 31;

    if (x[tid] != y[tid])
        atomicMin(&pos[threadIdx.y][threadIdx.z], tid);

    return x[pos[threadIdx.y][threadIdx.z]] >= y[pos[threadIdx.y][threadIdx.z]];
}

__device__ void cuSubtract(volatile uint32_t *x, volatile uint32_t *y, volatile uint32_t *z) {
    /* shared memory to hold underflow flags */
    __shared__ unsigned char s_borrow[BLOCK_DIM][BLOCK_DIM][32];
    unsigned char *borrow = s_borrow[threadIdx.y][threadIdx.z];
    int tid = threadIdx.x;

    /* set LSB's borrow to 0 */
    if (tid == 0)
        borrow[31] = 0;

    uint32_t t;
    t = x[tid] - y[tid];

    /* set the previous int's underflow flag if the subtraction answer is bigger than the subtractee */
    if (tid)
        borrow[tid - 1] = (t > x[tid]);

    /* keep processing until there's no flags */
    while (__any(borrow[tid])) {
        if (borrow[tid])
            t--;

        /* have to set flag if the new sub answer is 0xFFFFFFFF becuase of an underflow */
        if (tid)
            borrow[tid - 1] = (t == 0xFFFFFFFFu && borrow[tid]);
    }

    z[tid] = t;
}
