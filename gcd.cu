#include "hip/hip_runtime.h"
#include "gcd.h"

__device__ void shiftL1(bigInt num[]) {
   int flag = 0, flagn = 0;
	for (int i = 0; i < SIZE; i++) {
	   if (num[i] & HIGHBIT)
		   flagn = 1;
		
		num[i] <<= 1;
		if (flag)
		   num[i]++;
		
		flag = flagn;
		flagn = 0;
	}
}

__device__ void shiftR1(bigInt num[]) {
   int flag = 0, flagn = 0;
	for (int i = SIZE - 1; i >= 0; i--) {
	   if (num[i] & LOWBIT)
		   flagn = 1;
		
		num[i] >>= 1;
		if (flag)
		   num[i] |= HIGHBIT;
		
		flag = flagn;
		flagn = 0;
	}
}

// returns num1 (LT,EQ,GT)? num2
__device__ int cmp(bigInt num1[], bigInt num2[]) {
   for (int i = SIZE - 1; i >= 0; i--)
	   if (num1[i] != num2[i])
		   return (num1[i] < num2[i]) ? LT : GT;
	
	return EQ;
}

// requires that num1 >= num2, num1 -= num2
__device__ void cuSubtract(bigInt num1[], bigInt num2[]) {
   for (int i = 0; i < SIZE; i++) {
	   if (num2[i] <= num1[i]) {
		   // normal subtraction
			num1[i] = num1[i] - num2[i];
		} else {
		   // num1 - num2 == -1 * (num2 - num1)
			num1[i] = 1 + ~(num2[i] - num1[i]);
			
			if (num1[i+1] == 0)
			   num2[i+1]++;
			else
			   num1[i+1]--;
		}
	}
}

// eulers gcd algorithm without modulus
__device__ void slow_gcd(bigInt num1[], bigInt num2[]) {
   int compare;
	while ((compare = cmp(num1, num2)) != EQ) {
	   if (compare == GT)
		   cuSubtract(num1, num2);
		else
		   cuSubtract(num2, num1);
	}
}

// Binary GCD algorithm
// requires num1 > 0 and num2 > 0
// sets either num1 or num2 to the 1 if gcd == 1 or some number >1 if gcd >1 and
// returns the pointer to whichever num was set
__device__ bigInt* gcd(bigInt *num1, bigInt *num2) {
   int shift, compare;
	
	for (shift = 0; ((num1[0] | num2[0]) & LOWBIT) == 0; ++shift) {
	   shiftR1(num1);
		shiftR1(num2);
	}
	
	while ((num1[0] & 1) == 0)
	   shiftR1(num1);
	
	do {
	   while ((num2[0] & 1) == 0)
		   shiftR1(num2);
		
		compare = cmp(num1, num2);
		if (compare == EQ)
		   break;
		else if (compare == GT) {
		   bigInt *t = num1;
			num1 = num2;
			num2 = t;
		}
		cuSubtract(num2, num1);
	} while (1);
	
	if (shift)
	   shiftL1(num1);
	
	return num1;
}

__device__ bool greaterOne(bigInt *num) {
	for (int i = 0; i < SIZE; i++)
		if (i ? num[i] : num[i] > 1)
			return true;
	return false;
}

// count is the number of big nums in nums
// res represents a 2 dimensional matrix with at least count bits for each side
// should have count number of threads running, each responsible for 1 row/col
// res will be return as a top diagonal matrix
__global__ void findGCDs(bigInt *nums, int count, char *res, int offset) {	
   int ndx = blockIdx.x * blockDim.x + threadIdx.x; // == offset in bits
	int resOff = ndx * (1 + ((count - 1) / 8));

	bigInt cur[SIZE];
	bigInt other[SIZE];
	
   // do calc
   int i = ndx + offset + 1;
   int limit = min(i + WORK_SIZE, count);
	for (; i < limit; i++) {
		memcpy(cur, nums + ndx * SIZE, SIZE_BYTES);
		memcpy(other, nums + i * SIZE, SIZE_BYTES);
		
		if (greaterOne(gcd(cur, other)))
			res[resOff + i / 8] |= 1 << (i % 8);
	}
}
