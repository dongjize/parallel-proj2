#include "hip/hip_runtime.h"
#include "gcd.h"

__device__ void shiftL1(bigInt num[]) {
    int flag = 0, flagn = 0;
    for (int i = 0; i < SIZE; i++) {
        if (num[i] & HIGHBIT)
            flagn = 1;

        num[i] <<= 1;
        if (flag)
            num[i]++;

        flag = flagn;
        flagn = 0;
    }
}

__device__ void shiftR1(bigInt num[]) {
    int flag = 0, flagn = 0;
    for (int i = SIZE - 1; i >= 0; i--) {
        if (num[i] & LOWBIT)
            flagn = 1;

        num[i] >>= 1;
        if (flag)
            num[i] |= HIGHBIT;

        flag = flagn;
        flagn = 0;
    }
}

/**
 * returns num1 (LT,EQ,GT)? num2
 * @param num1
 * @param num2
 * @return
 */
__device__ int cmp(bigInt num1[], bigInt num2[]) {
    for (int i = SIZE - 1; i >= 0; i--)
        if (num1[i] != num2[i])
            return (num1[i] < num2[i]) ? LT : GT;

    return EQ;
}

/**
 * requires that num1 >= num2, num1 -= num2
 * @param num1
 * @param num2
 * @return
 */
__device__ void cuSubtract(bigInt num1[], bigInt num2[]) {
    for (int i = 0; i < SIZE; i++) {
        if (num2[i] <= num1[i]) {
            // normal subtraction
            num1[i] = num1[i] - num2[i];
        } else {
            // num1 - num2 == -1 * (num2 - num1)
            num1[i] = 1 + ~(num2[i] - num1[i]);

            if (num1[i + 1] == 0)
                num2[i + 1]++;
            else
                num1[i + 1]--;
        }
    }
}

/**
 * Euler's gcd algorithm without modulus
 * @param num1
 * @param num2
 * @return
 */
__device__ void slow_gcd(bigInt num1[], bigInt num2[]) {
    int compare;
    while ((compare = cmp(num1, num2)) != EQ) {
        if (compare == GT)
            cuSubtract(num1, num2);
        else
            cuSubtract(num2, num1);
    }
}


__device__ bigInt *gcd(bigInt *num1, bigInt *num2) {
    int shift, compare;

    for (shift = 0; ((num1[0] | num2[0]) & LOWBIT) == 0; ++shift) {
        shiftR1(num1);
        shiftR1(num2);
    }

    while ((num1[0] & 1) == 0) {
        shiftR1(num1);
    }
    do {
        while ((num2[0] & 1) == 0) {
            shiftR1(num2);
        }
        compare = cmp(num1, num2);
        if (compare == EQ) {
            break;
        } else if (compare == GT) {
            bigInt *t = num1;
            num1 = num2;
            num2 = t;
        }
        cuSubtract(num2, num1);
    } while (1);

    if (shift) {
        shiftL1(num1);
    }

    return num1;
}

__device__ bool greaterOne(bigInt *num) {
    for (int i = 0; i < SIZE; i++)
        if (i ? num[i] : num[i] > 1)
            return true;
    return false;
    }


/**
 * should have count number of threads running, each responsible for 1 row/col
 * res will be return as a top diagonal matrix
 * @param nums
 * @param count: the number of big nums in nums
 * @param res: represents a 2 dimensional matrix with at least count bits for each side
 * @param offset
 * @return
 */
__global__ void findGCDs(bigInt *nums, int count, char *res, int offset) {
    int ndx = blockIdx.x * blockDim.x + threadIdx.x; // == offset in bits
    int resOff = ndx * (1 + ((count - 1) / 8));
    bigInt cur[SIZE];
    bigInt other[SIZE];

    int i = ndx + offset + 1;
    int limit = min(i + WORK_SIZE, count);
    for (; i < limit; i++) {
        memcpy(cur, nums + ndx * SIZE, SIZE_BYTES);
        memcpy(other, nums + i * SIZE, SIZE_BYTES);

        if (greaterOne(gcd(cur, other))) {
            res[resOff + i / 8] |= 1 << (i % 8);
        }
    }
}


//__global__ void gmpGCDs(bigInt *nums, int count, char *res) {
//    mpz_t cur, other, g;
//    mpz_inits(cur, other, g, NULL);
//
//    for (int ndx = 0; ndx < count; ndx++) {
//        int resOff = ndx * (1 + ((count - 1) / 8));
//        mpz_import(cur, SIZE, -1, BIGINT_SIZE, -1, 0, nums + ndx * SIZE);
//
//        for (int i = ndx + 1; i < count; i++) {
//            mpz_import(other, SIZE, -1, BIGINT_SIZE, -1, 0, nums + i * SIZE);
//            mpz_gcd(g, cur, other);
//            if (mpz_cmp_ui(g, 1) > 0)
//                res[resOff + i / 8] |= 1 << (i % 8);
//        }
//    }
//}